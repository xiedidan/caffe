#include "hip/hip_runtime.h"
#include <algorithm>
#include <cfloat>
#include <vector>

#include "caffe/layers/dice_loss_layer.hpp"
#include "caffe/util/math_functions.hpp"

namespace caffe {
	template <typename Dtype>
	__global__ void ArgMax(const int n, const Dtype* data, Dtype* prediction) {
		CUDA_KERNEL_LOOP(i, n) {
			prediction[i] = data[i] >= data[i + n] ? 1 : 0;
		}
	}

	template <typename Dtype>
	__global__ void SegmentSum(const int count, const int segmentSize, const Dtype* data, Dtype* sum) {
		CUDA_KERNEL_LOOP(i, count) {
			// TODO : faster implementation?
			int sumIndex = i / segmentSize;
			sum[sumIndex] += data[i];
		}
	}

	template <typename Dtype>
	__global__ void gpuMemset(const int count, Dtype* data, const Dtype value) {
		CUDA_KERNEL_LOOP(i, count) {
			data[i] = value;
		}
	}

	template <typename Dtype>
	void DiceLossLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom, const vector<Blob<Dtype>*>& top) {
		const Dtype* data = bottom[0]->gpu_data();
		const Dtype* label = bottom[1]->gpu_data();

		const int labelCount = bottom[1]->count();
		const int batchSize = bottom[1]->num();
		const int dimSize = labelCount / batchSize;

		// call cuda method to compute prediction

		// NOLINT_NEXT_LINE(whitespace/operators)
		ArgMax<Dtype> <<<CAFFE_GET_BLOCKS(labelCount), CAFFE_CUDA_NUM_THREADS>>>(
			labelCount,
			data,
			bottom[1]->mutable_gpu_diff()
		);
		const Dtype* prediction = bottom[1]->gpu_diff();
		
		caffe_gpu_asum(labelCount, prediction, predictionSum.mutable_cpu_data());
		caffe_gpu_asum(labelCount, label, labelSum.mutable_cpu_data());
		caffe_gpu_mul(labelCount, prediction, label, bottom[1]->mutable_gpu_diff());
		caffe_gpu_asum(labelCount, bottom[1]->gpu_diff(), intersectionSum.mutable_cpu_data());
		top[0]->mutable_cpu_data()[0] = 2.0 * intersectionSum.cpu_data()[0] / (predictionSum.cpu_data()[0] + labelSum.cpu_data()[0]);
		
		/*
		// NOLINT_NEXT_LINE(whitespace/operators)
		gpuMemset<Dtype> << <CAFFE_GET_BLOCKS(batchSize), CAFFE_CUDA_NUM_THREADS >> >(
			batchSize,
			predictionSum.mutable_gpu_data(),
			Dtype(0)
		);

		// NOLINT_NEXT_LINE(whitespace/operators)
		SegmentSum<Dtype> <<<CAFFE_GET_BLOCKS(labelCount), CAFFE_CUDA_NUM_THREADS>>>(
			labelCount,
			dimSize,
			prediction,
			predictionSum.mutable_gpu_data()
		);

		// NOLINT_NEXT_LINE(whitespace/operators)
		gpuMemset<Dtype> << <CAFFE_GET_BLOCKS(batchSize), CAFFE_CUDA_NUM_THREADS >> >(
			batchSize,
			labelSum.mutable_gpu_data(),
			Dtype(0)
		);

		// NOLINT_NEXT_LINE(whitespace/operators)
		SegmentSum<Dtype> <<<CAFFE_GET_BLOCKS(labelCount), CAFFE_CUDA_NUM_THREADS>>>(
			labelCount,
			dimSize,
			label,
			labelSum.mutable_gpu_data()
		);
		
		caffe_gpu_mul(labelCount, prediction, label, bottom[1]->mutable_gpu_diff());

		// NOLINT_NEXT_LINE(whitespace/operators)
		gpuMemset<Dtype> << <CAFFE_GET_BLOCKS(batchSize), CAFFE_CUDA_NUM_THREADS >> >(
			batchSize,
			intersectionSum.mutable_gpu_data(),
			Dtype(0)
		);

		// NOLINT_NEXT_LINE(whitespace/operators)
		SegmentSum<Dtype> <<<CAFFE_GET_BLOCKS(labelCount), CAFFE_CUDA_NUM_THREADS>>>(
			labelCount,
			dimSize,
			bottom[1]->gpu_diff(),
			intersectionSum.mutable_gpu_data()
		);

		// total dice - it's simple so we directly compute on cpu
		top[0]->mutable_cpu_data()[0] = Dtype(0);
		for (int i = 0; i < batchSize; i++) {
			 // printf("i: %f, p: %f, l: %f\n", intersectionSum.cpu_data()[i], predictionSum.cpu_data()[i], labelSum.cpu_data()[i]);
			top[0]->mutable_cpu_data()[0] += 2.0 * intersectionSum.cpu_data()[i] / (predictionSum.cpu_data()[i] + labelSum.cpu_data()[i]);
		}
		*/
	}

	template <typename Dtype>
	__global__ void SegmentDiff(const int count, const Dtype* data, const Dtype* label, const Dtype* predictionSum, const Dtype* labelSum, const Dtype* intersectionSum, Dtype* diff) {
		CUDA_KERNEL_LOOP(i, count) {
			Dtype u = predictionSum[0] + labelSum[0];

			diff[i] = 2.0 * ((label[i] * u) / (u * u) - 2.0 * (data[i] * intersectionSum[0]) / (u * u));
			diff[i + count] = -2.0 * ((label[i] * u) / (u * u) - 2.0 * (data[i + count] * intersectionSum[0]) / (u * u));
		}
	}

	template <typename Dtype>
	void DiceLossLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top, const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
		if (propagate_down[1]) {
			LOG(FATAL) << this->type()
				<< " Layer cannot backpropagate to label inputs.";
		}

		if (propagate_down[0]) {
			const Dtype* data = bottom[0]->gpu_data();
			const Dtype* label = bottom[1]->gpu_data();

			const int labelCount = bottom[1]->count();
			const int batchSize = bottom[0]->num();
			const int dimSize = labelCount / batchSize;

			// NOLINT_NEXT_LINE(whitespace/operators)
			SegmentDiff<Dtype> <<<CAFFE_GET_BLOCKS(labelCount), CAFFE_CUDA_NUM_THREADS>>>(
				labelCount, 
				data, label,
				predictionSum.gpu_data(), labelSum.gpu_data(), intersectionSum.gpu_data(),
				bottom[0]->mutable_gpu_diff()
			);
		}
	}

	INSTANTIATE_LAYER_GPU_FUNCS(DiceLossLayer);

} // namespace caffe